#include "hip/hip_runtime.h"
#include "assign_color.cu"
#include "indept_set.cu"

#define NUM_COLOR_PER_THREAD 1 //this is changed to be more than one, then we need to move my_offset_start and my_offset_end to be in shared memory instead 

__device__ int numColored = 0;

__global__ void graphColoring(uint32_t NumRow, //number of vertices (= number of rows in adjacency matrix)
	                     uint32_t numNNZ, //number of non zero entry of the adjacency matrix
	                     uint32_t *col_id, //the column id in the CSR format 
	                     uint32_t *offset, //the row offset in the CSR
	                     int* color, //the color of the vertices (output)
	                     bool*set //the indepent set (global memory)
	                     ){

	int currentColor = 1;
	

	while(numColored < NumRow){//loop untill all vertices are colored 

		indept_set(NumRow, numNNZ, col_id, offset, set, currentColor%2 == 1, color, numColored);
		__syncthreads();		
		assign_color(currentColor, NumRow, set,color);
		__syncthreads();
		currentColor++;	
	}		
}
