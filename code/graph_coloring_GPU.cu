#include <sstream>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

//Error handling micro, wrap it around function whenever possible
static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("\n%s in %s at line %d\n", hipGetErrorString(err), file, line);
		//system("pause");
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#include "validate.h"
#include "serial.h"
#include "utility.h"
#include "coloring.cu"
#include "cuda_query.cu"
#include "GraphColoringKernel.h"

int main(int argc, char* argv[])
{
	cuda_query(0); //Set the deivde number here 

	if(argc != 2){
		std::cout<<"  Usage ./graphGPU INPUTFILE"<<std::endl;
		std::cout<<"input files can be found under input/ "<<std::endl;
		exit(EXIT_FAILURE);
	}

   bool* graph;
   int V;  
   uint32_t numNNZ=0;
   uint32_t NumRow=0; //same as V


   //1) Read graph
   if (std::string(argv[1]).find(".col") != std::string::npos){
     ReadColFile(argv[1], &graph, &V, &numNNZ,&NumRow);
   } else if (std::string(argv[1]).find(".mm") != std::string::npos){
     ReadMMFile(argv[1], &graph, &V, &numNNZ,&NumRow);
   } else{
   	std::cout<<" Invalid file formate!!"<<std::endl;
   	exit(EXIT_FAILURE);
   }
   /***********************************************************************/

   //2) Allocate memory (on both sides)
   uint32_t *col_id(NULL),*offset(NULL);   
   HANDLE_ERROR(hipMallocManaged(&col_id, numNNZ*sizeof(uint32_t)));
   
   //last entry will be = numNonZero (so that we have always a pointer
   //to the first and last id for each row with no need for if statments)   
   HANDLE_ERROR(hipMallocManaged(&offset, (NumRow +1)*sizeof(uint32_t)));
   /***********************************************************************/

   //3) Get graph in CSR format 
   getCSR(numNNZ, NumRow, graph, col_id, offset);
   //printCSR(numNNZ,NumRow,col_id, offset);
   /***********************************************************************/

   //5) Color Vertices in paralllel
   int* color;
   HANDLE_ERROR(hipMallocManaged(&color, NumRow*sizeof(int)));
   memset(color, 0, NumRow );   

   bool*set;
   HANDLE_ERROR(hipMallocManaged(&set, NumRow*sizeof(bool)));
   memset(set, 0, NumRow); 

   coloring(NumRow, numNNZ, col_id, offset, color, set);
   
   //6) Validate parallel solution 
   printf("Parallel solution has %d colors\n", CountColors(V, color));
   printf("Valid coloring: %d\n\n", IsValidColoring(graph, V, color));
   //PrintSolution(color,V);
   /***********************************************************************/


   //7) Color Vertices on CPU
   GraphColoring(graph, V, &color);
   printf("Brute-foce solution has %d colors\n", CountColors(V, color));   
   printf("Valid coloring: %d\n", IsValidColoring(graph, V, color));

   GreedyColoring(graph, V, &color);
   printf("\n***************\n");
   printf("Greedy solution has %d colors\n", CountColors(V, color));
   printf("Valid coloring: %d\n\n", IsValidColoring(graph, V, color));
   //PrintSolution(color,V);
   /***********************************************************************/


   //8)GraphColoring:let each thread compare one host vertex's value with one of its neighbor vertexes
   standard_context_t context;
   uint32_t sizeNode = NumRow;
   uint32_t sizeLbs = numNNZ;
   int blockSize = 256;
   int gridSize = sizeLbs / blockSize + 1;
   int* lbs;
   int* wir;
   HANDLE_ERROR(hipMallocManaged(&lbs, numNNZ*sizeof(int)));
   HANDLE_ERROR(hipMallocManaged(&wir, numNNZ*sizeof(int)));
   load_balance_search(sizeLbs, (int*)offset, sizeNode,lbs,context);
   hipDeviceSynchronize();
   WorkItemRank<<<gridSize,blockSize>>>((int*)offset, lbs, wir, sizeLbs);
   hipDeviceSynchronize();

   for(int i = 0 ; i < V; i++)
      {
          randoms[i] = i;
       }

   bool* setTrue;
   HANDLE_ERROR(hipMallocManaged(&setTrue, NumRow*sizeof(bool)));

   for(int c = 1; c < 254; c++)
   {
        int threadnum = 256;
        int blocknum = V / threadnum + 1;
        memset(setTrue, true, NumRow); 
        GraphColoringKernel<<<blocknum,threadnum>>>(c, NumRow, col_id, offset, lbs, wir, randoms, color, setTrue);
        hipDeviceSynchronize();
        ColorChanging<<blocknum,threadnum>>(c, NumRow, color, setTrue);
        hipDeviceSynchronize();
    }

   printf("GraphColoringKernel found solution with %d colors\n", CountColors(V, color));
   printf("Valid coloring: %d\n", IsValidColoring(graph, V, color));
   /***********************************************************************/

   return 0;
}
